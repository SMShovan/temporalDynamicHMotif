#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <climits>
#include <hip/hip_runtime.h>

#include "../include/graphGeneration.hpp"
#include "../include/utils.hpp"
#include "../include/structure.hpp"

// Local helper mirroring main.cu's flatten2DVector logic with sentinel padding to multiple of 4
static std::pair<std::vector<int>, std::vector<int>> localFlatten2D(const std::vector<std::vector<int>>& vec2d) {
    auto nextMultipleOf4 = [](int x) { return ((x + 3) / 4) * 4; };
    std::vector<int> flatValues;
    std::vector<int> startOffsets(vec2d.size());
    int cursor = 0;
    for (size_t i = 0; i < vec2d.size(); ++i) {
        startOffsets[i] = cursor;
        int inner = static_cast<int>(vec2d[i].size());
        int padded = nextMultipleOf4(inner);
        for (int j = 0; j < padded; ++j) {
            if (j < inner) flatValues.push_back(vec2d[i][j]);
            else if (j == padded - 1) flatValues.push_back(INT_MIN);
            else flatValues.push_back(0);
            ++cursor;
        }
    }
    return {flatValues, startOffsets};
}

__global__ void countType1PerHyperedgeKernel(const int* __restrict__ startOffsets,
                                             const int* __restrict__ flatValues,
                                             long long* __restrict__ outCounts,
                                             int numHyperedges,
                                             int fixedSize) {
    int e = blockIdx.x * blockDim.x + threadIdx.x;
    if (e >= numHyperedges) return;
    int start = startOffsets[e];
    // compute degree by scanning until INT_MIN or 0
    int n = 0;
    int pos = start;
    while (pos < fixedSize) {
        int v = flatValues[pos];
        if (v == INT_MIN || v == 0) break;
        ++n; ++pos;
    }
    long long c = 0;
    if (n >= 3) {
        long long nn = n;
        c = (nn * (nn - 1) * (nn - 2)) / 6;
    }
    outCounts[e] = c;
}

static inline void checkCudaLocal(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        std::exit(-1);
    }
}

int main(int argc, char* argv[]) {
    // Parameters: <num_hyperedges> <max_vertices_per_hyperedge> <min_vertex_id> <max_vertex_id> <payload_capacity> [alignment=4]
    HypergraphParams params;
    if (!parseCommandLineArgs(argc, argv, params)) {
        return 1;
    }
    printHypergraphParams(params);

    // Generate host-side mappings
    auto [hyperedgeToVertex, vertexToHyperedge] = generateHypergraph(params);

    // Flatten H2V locally and prepare CBST inputs
    auto [h2vFlatValues, h2vStartOffsets] = localFlatten2D(hyperedgeToVertex);

    // Prepare CBST key/start arrays
    std::vector<int> h2vKeys(params.numHyperedges);
    for (int i = 0; i < params.numHyperedges; ++i) h2vKeys[i] = i + 1;

    // Build H2V CBST to allocate device buffers and copy payload
    CBSTOperations h2vOps("H2V", params.payloadCapacity, params.alignment);
    h2vOps.construct(h2vKeys.data(), h2vStartOffsets.data(), params.numHyperedges,
                     h2vFlatValues.data(), static_cast<int>(h2vFlatValues.size()));

    const CBSTContext& ctx = h2vOps.context();

    // Allocate output array on device
    long long* d_counts = nullptr;
    checkCudaLocal(hipMalloc(&d_counts, params.numHyperedges * sizeof(long long)));
    checkCudaLocal(hipMemset(d_counts, 0, params.numHyperedges * sizeof(long long)));

    // Launch kernel
    int block = 256;
    int grid = (params.numHyperedges + block - 1) / block;
    countType1PerHyperedgeKernel<<<grid, block>>>(ctx.d_startOffsets, ctx.d_flatPayload, d_counts, params.numHyperedges, ctx.fixedSize);
    checkCudaLocal(hipDeviceSynchronize());

    // Copy back and reduce on host
    std::vector<long long> counts(params.numHyperedges);
    checkCudaLocal(hipMemcpy(counts.data(), d_counts, params.numHyperedges * sizeof(long long), hipMemcpyDeviceToHost));
    long long total = 0;
    for (long long c : counts) total += c;
    std::cout << "[GPU] Type1 motif count (sum_e C(|e|, 3)): " << total << std::endl;

    checkCudaLocal(hipFree(d_counts));
    return 0;
}


